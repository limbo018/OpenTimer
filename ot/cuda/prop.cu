#include "hip/hip_runtime.h"
/**
 * @file   prop.cu
 * @author Yibo Lin
 * @date   Mar 2020
 */

#include <cstdio>
#include <ot/cuda/prop.cuh>
#include "ot/cuda/utils.cuh"

enum Split {
  MIN = 0,
  MAX = 1
};

enum Tran {
  RISE = 0,
  FALL = 1
};

#define MAX_SPLIT_TRAN 4 
#define MAX_SPLIT 2
#define MAX_TRAN 2

template <typename T>
void print(const T* data, int n, const char* msg) {
    printf("%s[%d] = {", msg, n);
    for (int i = 0; i < n; ++i) {
        printf("%g ", (double)data[i]);
    }
    printf("}\n");
}

void print(FlatTableCUDA const& ft, const char* msg) {
    printf("%s[%u] = {\n", msg, ft.num_tables);
    for (int i = 0; i < ft.num_tables; ++i) {
        printf("slew[%u][%u x %u]\n", i, ft.xs_st[i + 1] - ft.xs_st[i], ft.ys_st[i + 1] - ft.ys_st[i]);
        printf("ft.xs: ");
        for (int j = ft.xs_st[i]; j < ft.xs_st[i + 1]; ++j) {
            printf("%g ", ft.xs[j]); 
        }
        printf("\n");
        printf("ft.ys: ");
        for (int j = ft.ys_st[i]; j < ft.ys_st[i + 1]; ++j) {
            printf("%g ", ft.ys[j]); 
        }
        printf("\n");
        printf("slew table\n");
        for (int j = ft.data_st[i]; j < ft.data_st[i + 1]; ++j) {
            printf("%g ", ft.data[j]); 
            if ((j % (ft.ys_st[i + 1] - ft.ys_st[i])) == (ft.ys_st[i + 1] - ft.ys_st[i] - 1)) {
                printf("\n");
            }
        }
        printf("\n");
    }
    printf("}\n");
}

void PropCUDA::destroy_device() {
    fanin_graph.destroy_device();
    slew_ft.destroy_device(); 
    delay_ft.destroy_device();

    destroyCUDA(net_arc_delays); 
    destroyCUDA(net_arc_impulses); 
    destroyCUDA(pin_loads); 
    destroyCUDA(arc2ftid); 
    destroyCUDA(frontiers); 
    destroyCUDA(frontiers_ends); 
    destroyCUDA(pin_slews); 
    destroyCUDA(pin_ats); 
    destroyCUDA(cell_arc_delays); 
}

void PropCUDA::copy2device(PropCUDA& rhs) const {
    fanin_graph.copy2device(rhs.fanin_graph); 
    slew_ft.copy2device(rhs.slew_ft); 
    delay_ft.copy2device(rhs.delay_ft); 

    rhs.num_levels = num_levels; 
    rhs.num_pins = num_pins; 
    rhs.num_arcs = num_arcs; 
    allocateCopyCUDA(rhs.net_arc_delays, net_arc_delays, num_arcs * MAX_SPLIT_TRAN);
    allocateCopyCUDA(rhs.net_arc_impulses, net_arc_impulses, num_arcs * MAX_SPLIT_TRAN);
    allocateCopyCUDA(rhs.pin_loads, pin_loads, num_arcs * MAX_SPLIT_TRAN);
    allocateCopyCUDA(rhs.arc2ftid, arc2ftid, num_arcs * MAX_SPLIT_TRAN * MAX_TRAN);
    allocateCopyCUDA(rhs.frontiers, frontiers, num_pins);
    allocateCopyCUDA(rhs.frontiers_ends, frontiers_ends, num_levels + 1);
    allocateCopyCUDA(rhs.pin_slews, pin_slews, num_pins * MAX_SPLIT_TRAN); 
    allocateCopyCUDA(rhs.pin_ats, pin_ats, num_pins * MAX_SPLIT_TRAN); 
    allocateCopyCUDA(rhs.cell_arc_delays, cell_arc_delays, num_arcs * MAX_SPLIT_TRAN * MAX_TRAN);
}

__device__ float interpolate(float x1, float x2, float d1, float d2, float x) {
    if (x1 == x2) {
        return d1; 
    }
    else {
        return d1 + (d2 - d1) * (x - x1) / (x2 - x1);
    }
}

__device__ float lut_lookup(int n, int m, 
        const float *xs, const float *ys, const float *data,
        float x, float y) {
#define AT_DATA_2D(i, j) data[(i) * m + (j)]

  int i_1 = 0; 
  int i = min(1, n - 1);
  while(i + 1 < n && xs[i] <= x) {
      i_1 = i++;
  }
  int j_1 = 0; 
  int j = min(1, m - 1);
  while(j + 1 < m && ys[j] <= y) {
      j_1 = j++;
  }
  float r1 = interpolate(ys[j_1], ys[j], AT_DATA_2D(i_1, j_1), AT_DATA_2D(i_1, j), y);
  float r2 = interpolate(ys[j_1], ys[j], AT_DATA_2D(i, j_1), AT_DATA_2D(i, j), y);
  float r = interpolate(xs[i_1], xs[i], r1, r2, x);

#undef AT_DATA_2D
  return r;
}

__device__ float lut_lookup(const FlatTableCUDA &ft, int lutidx, float x, float y) {
  int xsl = ft.xs_st[lutidx], xsr = ft.xs_st[lutidx + 1];
  int ysl = ft.ys_st[lutidx], ysr = ft.ys_st[lutidx + 1];
  int datal = ft.data_st[lutidx];
  return lut_lookup(xsr - xsl, ysr - ysl, 
          ft.xs + xsl, ft.ys + ysl, ft.data + datal, 
          x, y);
}

__device__ int ftid(PropCUDA const& prop, int arc_id, int el, int irf, int orf) {
    int id = arc_id * (MAX_SPLIT_TRAN * MAX_TRAN) + el * (MAX_TRAN * MAX_TRAN)
        + irf * MAX_TRAN + orf; 
    return prop.arc2ftid[id]; 
}

__device__ void update_slew_or_at(PinInfoCUDA& to_slew, FlatArc const& arc, int el, int rf, float slew) {
    if (el == Split::MAX) {
        if (to_slew.value < slew) {
            to_slew.value = slew; 
            to_slew.from_el = el;
            to_slew.from_rf = rf; 
            to_slew.from_arcidx = (arc.idx >> 1);
        }
    } 
    else {
        if (to_slew.value > slew) {
            to_slew.value = slew; 
            to_slew.from_el = el;
            to_slew.from_rf = rf; 
            to_slew.from_arcidx = (arc.idx >> 1);
        }
    }
}

__global__ void fprop_slew_cuda(PropCUDA prop, int level_l, int level_r) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int offset = tid + level_l; 
    if (offset < level_r) {
        int el_trf = threadIdx.y; // trf is short for rf for to 
        int el = (el_trf >> 1);
        int trf = (el_trf & 1); 
        int pin_to = prop.frontiers[offset];
        int pin_to_offset = pin_to * MAX_SPLIT_TRAN + el_trf;
        auto to_load = prop.pin_loads[pin_to_offset];
        auto& to_slew = prop.pin_slews[pin_to_offset];
        auto& to_at = prop.pin_ats[pin_to_offset];
        int bgn = prop.fanin_graph.adjacency_list_start[pin_to]; 
        int end = prop.fanin_graph.adjacency_list_start[pin_to + 1]; 
        for (int e = bgn; e < end; ++e) {
            FlatArc const& arc = prop.fanin_graph.adjacency_list[e]; 
            int arc_idx = (arc.idx >> 1);
            int pin_from = arc.other;
            auto arc_type = (arc.idx & 1); 
            if (arc_type) { // cell arc 
                int pin_from_offset = pin_from * MAX_SPLIT_TRAN + el * MAX_TRAN;
                int arc_offset = arc_idx * MAX_SPLIT_TRAN * MAX_TRAN + el * MAX_TRAN * MAX_TRAN + trf; 
                for (int frf = 0; frf < MAX_TRAN; ++frf) {
                    auto const& from_slew = prop.pin_slews[pin_from_offset + frf]; 
                    auto const& from_at = prop.pin_ats[pin_from_offset + frf]; 
                    auto& arc_delay = prop.cell_arc_delays[arc_offset + frf * MAX_TRAN];
                    int lutidx = ftid(prop, arc_idx, el, frf, trf); 
                    if (lutidx < prop.slew_ft.num_tables) {
                        float cur_to_slew = lut_lookup(prop.slew_ft, lutidx, from_slew.value, to_load);
                        float delay = lut_lookup(prop.delay_ft, lutidx, from_slew.value, to_load);
                        float cur_to_at = from_at.value + delay;
                        arc_delay = delay; 
                        update_slew_or_at(to_slew, arc, el, trf, cur_to_slew);
                        update_slew_or_at(to_at, arc, el, trf, cur_to_at);
                    }
                }
            }
            else { // net arc 
                int pin_from_offset = pin_from * MAX_SPLIT_TRAN + el_trf;
                auto const& from_slew = prop.pin_slews[pin_from_offset];
                auto const& from_at = prop.pin_ats[pin_from_offset];
                int arc_offset = arc_idx * MAX_SPLIT_TRAN + el_trf;
                float arc_impulse = prop.net_arc_impulses[arc_offset];
                float arc_delay = prop.net_arc_delays[arc_offset];
                int sign = (from_slew.value < 0? -1 : 1);
                float cur_to_slew = sqrtf(from_slew.value * from_slew.value + arc_impulse) * sign; 
                float cur_to_at = from_at.value + arc_delay; 
                update_slew_or_at(to_slew, arc, el, trf, cur_to_slew);
                update_slew_or_at(to_at, arc, el, trf, cur_to_at);
            }
        }
    }
}

__global__ void print(PinInfoCUDA const* pin_slews, int n) {
    if (blockIdx.x == 0 && threadIdx.x == 0) {
        for (int i = 0; i < n; ++i) {
            for (int el = 0; el < MAX_SPLIT; ++el) {
                for (int rf = 0; rf < MAX_TRAN; ++rf) {
                    int idx = i * MAX_SPLIT_TRAN + el * MAX_TRAN + rf;
                    printf("data_cuda.pin[%u][%d][%d] slew %.6f\n", i, el, rf, pin_slews[idx].value);
                }
            }
        }
    }
}

void prop_cuda(PropCUDA& data_cpu) {
    //print(prop_data_cpu.arcs, prop_data_cpu.num_arcs); 
    //print(prop_data_cpu.net_arc_delays, prop_data_cpu.num_arcs, "net_arc_delays"); 
    //print(prop_data_cpu.net_arc_impulses, prop_data_cpu.num_arcs, "net_arc_impulses"); 
    //print(prop_data_cpu.pin_loads, prop_data_cpu.num_pins, "pin_loads"); 
    //print(prop_data_cpu.arc2ftid, prop_data_cpu.num_arcs, "arc2ftid"); 
    //print(prop_data_cpu.slew_ft, "slew_ft");
    //print(prop_data_cpu.delay_ft, "delay_ft");
    // kernel propagation  
    PropCUDA data_cuda;
    data_cpu.copy2device(data_cuda);

    constexpr int chunk = 32; 
    for (int i = data_cpu.num_levels - 1; i >= 0; --i) {
        int l = data_cpu.frontiers_ends[i]; 
        int r = data_cpu.frontiers_ends[i + 1];
        //printf("frontiers[%d]: ", data_cpu.num_levels - 1 - i);
        //for (int k = l; k < r; ++k) {
        //    printf("%d, ", data_cpu.frontiers[k]);
        //}
        //printf("\n");
        int block_dim = (r - l + chunk - 1) / chunk;
        fprop_slew_cuda<<<block_dim, {chunk, MAX_SPLIT_TRAN}>>>(data_cuda, l, r); 
        //fprop_at_cuda<<<block_dim, {chunk, MAX_SPLIT_TRAN}>>>(data_cuda, l, r); 
    }

    //print<<<1, 1>>>(data_cuda.pin_slews, data_cuda.num_pins); 

    checkCUDA(hipMemcpy(data_cpu.pin_slews, data_cuda.pin_slews, sizeof(PinInfoCUDA) * data_cpu.num_pins * MAX_SPLIT_TRAN, hipMemcpyDeviceToHost));
    checkCUDA(hipMemcpy(data_cpu.pin_ats, data_cuda.pin_ats, sizeof(PinInfoCUDA) * data_cpu.num_pins * MAX_SPLIT_TRAN, hipMemcpyDeviceToHost));
    checkCUDA(hipMemcpy(data_cpu.cell_arc_delays, data_cuda.cell_arc_delays, sizeof(float) * data_cpu.num_arcs * MAX_SPLIT_TRAN * MAX_TRAN, hipMemcpyDeviceToHost));
    checkCUDA(hipDeviceSynchronize());

    //for (int i = 0; i < data_cpu.num_pins; ++i) {
    //    for (int el = 0; el < MAX_SPLIT; ++el) {
    //        for (int rf = 0; rf < MAX_TRAN; ++rf) {
    //            int idx = i * MAX_SPLIT_TRAN + el * MAX_TRAN + rf;
    //            printf("data_cpu.pin[%u][%d][%d] slew %.6f\n", i, el, rf, data_cpu.pin_slews[idx].value);
    //        }
    //    }
    //}
    data_cuda.destroy_device(); 
}
