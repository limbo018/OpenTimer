/*
 * allocator wrapper for cudaMallocHost that can be used in STL containers.
 * @author Zizheng Guo
 * reference: https://en.cppreference.com/w/cpp/named_req/Allocator
 */


#include <hip/hip_runtime.h>
#include <limits>
#include <memory>
#include <cassert>

void *wrapped_cudaMallocHost(std::size_t n) {
  void *ret;
  hipError_t status = hipHostMalloc(&ret, n, hipHostMallocDefault);
  if(status != hipSuccess) return NULL;
  else return ret;
}

void wrapped_cudaFreeHost(void *ptr) {
  assert(hipSuccess == hipHostFree(ptr));
}
