#include "hip/hip_runtime.h"
//#include <ot/cuda/toposort.cuh>
#include <ot/cuda/prop.cuh>
#include <ot/cuda/utils.cuh>

__global__ void toposort_advance(int *edgelist_start, FlatArc *edgelist,
                                 int *out,
                                 int *frontiers,
                                 int last_size, int *new_size) {
  int id = blockIdx.x * blockDim.x + threadIdx.x;
  if(id >= last_size) return;
  
  int *next_frontiers = frontiers + last_size;
  int u = frontiers[id];
  int edge_st = edgelist_start[u], edge_ed = edgelist_start[u + 1];
  
  for(int i = edge_st; i < edge_ed; ++i) {
    int v = edgelist[i].other;
    if(1 == atomicAdd(&out[v], -1)) {
      next_frontiers[atomicAdd(new_size, 1)] = v;
    }
  }
}

const int thread_per_block = 128;

void toposort_compute_cuda(
  int first_size, PropCUDA& prop_data_cpu, PropCUDA& prop_data_cuda, 
  std::vector<int> &frontiers_ends)
{
    int n = prop_data_cuda.fanin_graph.num_nodes; 
    int num_edges = prop_data_cuda.fanin_graph.num_edges;
  // Step 1: copy to GPU
  int *new_size_gpu;

  allocateCUDA(new_size_gpu, 1, int);
  checkCUDA(hipDeviceSynchronize());

  // Step 2: do the computation
  int total_size_wo_last = 0, last_size = first_size;
  while(true) {
    toposort_advance<<<(last_size + thread_per_block - 1) / thread_per_block,
      thread_per_block>>>(prop_data_cuda.fanin_graph.adjacency_list_start, prop_data_cuda.fanin_graph.adjacency_list, prop_data_cuda.fanout_degrees,
                          prop_data_cuda.frontiers + total_size_wo_last,
                          last_size, new_size_gpu);
      checkCUDA(hipDeviceSynchronize());
    int current_size;
    memcpyDeviceHostCUDA(&current_size, new_size_gpu, 1);
    checkCUDA(hipDeviceSynchronize());
    if(current_size) {
      total_size_wo_last += last_size;
      last_size = current_size;
      frontiers_ends.push_back(total_size_wo_last + current_size);
      
      current_size = 0;
      memcpyHostDeviceCUDA(new_size_gpu, &current_size, 1);
      checkCUDA(hipDeviceSynchronize());
    }
    else break;
  }

  // Step 3: copy back to CPU
  memcpyDeviceHostCUDA(prop_data_cpu.frontiers, prop_data_cuda.frontiers, n);
  destroyCUDA(new_size_gpu);
  checkCUDA(hipDeviceSynchronize());
}
